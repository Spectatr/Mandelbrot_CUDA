
#include <hip/hip_runtime.h>
#include <X11/Xlib.h>
#include <X11/Xutil.h>
#include <X11/Xos.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#define		X_RESN	800       /* x resolution */
#define		Y_RESN	800       /* y resolution */

typedef struct complextype
	{
        float real, imag;
	} Compl;

__global__ void calc(int* pixels);

int main ()
{
	Window		win;                            /* initialization for a window */
	unsigned
	int             width, height,                  /* window size */
                        x, y,                           /* window position */
                        border_width,                   /*border width in pixels */
                        display_width, display_height,  /* size of screen */
                        screen;                         /* which screen */

	char            *window_name = "Mandelbrot Set", *display_name = NULL;
	GC              gc;
	unsigned
	long		valuemask = 0;
	XGCValues	values;
	Display		*display;
	XSizeHints	size_hints;
	Pixmap		bitmap;
	XPoint		points[800];
	FILE		*fp, *fopen ();
	char		str[100];
	
	XSetWindowAttributes attr[1];

       /* Mandlebrot variables */
        int i, j, k;
        
        
       
	/* connect to Xserver */

	if (  (display = XOpenDisplay (display_name)) == NULL ) {
	   fprintf (stderr, "drawon: cannot connect to X server %s\n",
				XDisplayName (display_name) );
	exit (-1);
	}
	
	/* get screen size */

	screen = DefaultScreen (display);
	display_width = DisplayWidth (display, screen);
	display_height = DisplayHeight (display, screen);

	/* set window size */

	width = X_RESN;
	height = Y_RESN;

	/* set window position */

	x = 0;
	y = 0;

        /* create opaque window */

	border_width = 4;
	win = XCreateSimpleWindow (display, RootWindow (display, screen),
				x, y, width, height, border_width, 
				BlackPixel (display, screen), WhitePixel (display, screen));

	size_hints.flags = USPosition|USSize;
	size_hints.x = x;
	size_hints.y = y;
	size_hints.width = width;
	size_hints.height = height;
	size_hints.min_width = 300;
	size_hints.min_height = 300;
	
	XSetNormalHints (display, win, &size_hints);
	XStoreName(display, win, window_name);

        /* create graphics context */

	gc = XCreateGC (display, win, valuemask, &values);

	XSetBackground (display, gc, WhitePixel (display, screen));
	XSetForeground (display, gc, BlackPixel (display, screen));
	XSetLineAttributes (display, gc, 1, LineSolid, CapRound, JoinRound);

	attr[0].backing_store = Always;
	attr[0].backing_planes = 1;
	attr[0].backing_pixel = BlackPixel(display, screen);

	XChangeWindowAttributes(display, win, CWBackingStore | CWBackingPlanes | CWBackingPixel, attr);

	XMapWindow (display, win);
	XSync(display, 0);

        /* Calculate and draw points */

dim3 blockDim(16,16);
dim3 gridDim(800/blockDim.x,800/blockDim.y);
int* pixels;
int pixels2[800*800];

hipMalloc((void**)&pixels,800*800*sizeof(int));

//timer
double gpuTime;
unsigned int hTimer;

hipEvent_t start, stop;
float time,time2=0;
int h;

hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord( start, 0 );
calc<<<gridDim, blockDim>>>(pixels);
hipEventRecord( stop, 0 );
hipEventSynchronize( stop );
hipEventElapsedTime( &time, start, stop );
time2+=time;
hipEventDestroy( start );
hipEventDestroy( stop );

printf("TIME: %f\n",time2/1000);
//

hipMemcpy(pixels2,pixels,800*800*sizeof(int),hipMemcpyDeviceToHost);

for(i=0;i<800;i++){
for(j=0;j<800;j++){
	if(pixels2[i*800+j]==1){
		XDrawPoint (display, win, gc, j, i);
	}
}}

	XFlush (display);
	hipFree(pixels);
	sleep (1);

	/* Program Finished */

}

__global__ void calc(int* pixels){
Compl	z, c;
int i,j,k;
float	lengthsq, temp;

i=blockIdx.x*blockDim.x+threadIdx.x;
j=blockIdx.y*blockDim.y+threadIdx.y;



          z.real = z.imag = 0.0;
          c.real = ((float) j - 400.0)/200.0;               /* scale factors for 800 x 800 window */
	  c.imag = ((float) i - 400.0)/200.0;
          k = 0;

          do  {                                             /* iterate for pixel color */

            temp = z.real*z.real - z.imag*z.imag + c.real;
            z.imag = 2.0*z.real*z.imag + c.imag;
            z.real = temp;
            lengthsq = z.real*z.real+z.imag*z.imag;
            k++;

          } while (lengthsq < 4.0 && k < 100);

        if (k == 100){
	pixels[i*800+j]=1;
        }
}